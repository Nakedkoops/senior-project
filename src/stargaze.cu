#include "hip/hip_runtime.h"
#define WIEN_B 28980000

#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>

namespace py = pybind11;

const double FRAUNHOFER_LINES[] = {
    898.765,  // O2
    822.696,  // O2
    759.370,  // O2
    686.719,  // O2
    656.281,  // H
    627.661,  // O2
    589.592,  // Na
    588.995,  // Na
    587.5618, // He
    546.073,  // Hg
    527.039,  // Fe
    518.362,  // Mg
    517.270,  // Mg
    516.891,  // Fe
    516.733,  // Mg
    495.761,  // Fe
    486.134,  // H
    466.814,  // Fe
    438.355,  // Fe
    434.047,  // H
    430.790,  // Fe
    430.774,  // Ca
    410.175,  // H
    396.847,  // Ca+
    393.368,  // Ca+
    382.044,  // Fe
    358.121,  // Fe
    336.112,  // Ti+
    302.108,  // Fe
    299.444,  // Ni
};

struct max_flux_idx {
    __device__
    thrust::tuple<size_t, double> operator()(const thrust::tuple<size_t, double> &a, const thrust::tuple<size_t, double> &b) {
        return thrust::get<1>(b) > thrust::get<1>(a) ? b : a;
    }
};

struct get_temperature {
    size_t samples_per_spectra;
    float first_wavelength;
    float dispersion_per_pixel;

    __host__
    get_temperature(size_t _samples_per_spectra, float _first_wavelength, float _dispersion_per_pixel):
        samples_per_spectra(_samples_per_spectra),
        first_wavelength(_first_wavelength),
        dispersion_per_pixel(_dispersion_per_pixel) {}

    __device__
    float operator()(size_t star_idx, size_t idx) const {
        size_t offset = idx - star_idx * samples_per_spectra;
        float wavelength = __exp10f(first_wavelength + offset * dispersion_per_pixel);

        return WIEN_B / wavelength;
    }
};

// Calculate temperatures by using Wien's displacement law:
//
//      T = b / λ_peak
//
// where `b` is Wien's displacement constant, equal to
//
//      28,980,000 Å*K
//
// The parameter type `py::array::c_style | py::array::forcecast` restricts this to only
// accept "dense" arrays that we can directly reinterpret as a row-major `double*`
//
// https://pybind11.readthedocs.io/en/stable/advanced/pycpp/numpy.html#arrays
py::array_t<float> temperatures(
    py::array_t<double, py::array::c_style | py::array::forcecast> py_model,
    float first_wavelength,
    float dispersion_per_pixel
) {
    py::buffer_info buf_model = py_model.request();

    size_t spectra_per_run;
    size_t samples_per_spectra;
    size_t buf_size = buf_model.size;

    if (buf_model.ndim == 2) {
        spectra_per_run = buf_model.shape[0];
        samples_per_spectra = buf_model.shape[1];
    } else {
        spectra_per_run = 1;
        samples_per_spectra = buf_model.shape[0];
    }

    double* model = reinterpret_cast<double*>(buf_model.ptr);
    thrust::device_vector<double> d_model(model, model + buf_size);

    // The star index will act as our key in the following reduction,
    // since we want to get the highest-flux wavelength for EACH star.

    auto idx_begin = thrust::make_counting_iterator<size_t>(0);
    auto idx_end = thrust::make_counting_iterator<size_t>(buf_size);

    auto idx_star_begin = thrust::make_transform_iterator(idx_begin, thrust::placeholders::_1 / samples_per_spectra);
    auto idx_star_end = thrust::make_transform_iterator(idx_end, thrust::placeholders::_1 / samples_per_spectra);

    auto idx_sample_begin = thrust::make_transform_iterator(idx_begin, thrust::placeholders::_1 % samples_per_spectra);

    // First we find the index where the maximum flux occurs
    thrust::device_vector<size_t> d_max_flux_idx(spectra_per_run);
    thrust::reduce_by_key(
        idx_star_begin,
        idx_star_end,
        thrust::make_zip_iterator(thrust::make_tuple(
            idx_sample_begin,
            d_model.begin()
        )),
        thrust::make_discard_iterator(), // We don't care about the index of the star in the output
        thrust::make_zip_iterator(thrust::make_tuple(
            d_max_flux_idx.begin(), 
            thrust::make_discard_iterator() // We don't care about the value of the peak
        )),
        thrust::equal_to<size_t>(),
        max_flux_idx()
    );

    thrust::device_vector<float> d_temperatures(spectra_per_run);
    thrust::transform(
        idx_star_begin,
        idx_star_end,
        d_max_flux_idx.begin(),
        d_temperatures.begin(),
        get_temperature(samples_per_spectra, first_wavelength, dispersion_per_pixel)
    );

    thrust::host_vector<float> temperatures(d_temperatures);

    return py::array_t<float>(
        { spectra_per_run },
        { sizeof(float) },
        thrust::raw_pointer_cast(temperatures.data())
    );
}

// Define the Python FFI bindings
PYBIND11_MODULE(stargaze, m)
{
    m.doc() = "";
    m.def("temperatures", temperatures);
}
